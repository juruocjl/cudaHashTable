#include "hip/hip_runtime.h"
#include "dinner123.h"
#include "hash_table.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <map>
size_t size(){
  return 0;
}
size_t capacity(){
  return 0;
}
const int MultiProcessorCount = 80;
#define mix(h) ({					\
			(h) ^= (h) >> 23;		\
			(h) *= 0x2127599bf4325c37ULL;	\
			(h) ^= (h) >> 47; })


uint64_t fasthash64(uint64_t v) {
	const uint64_t m = 0x880355f21e6d1965ULL;
	uint64_t h = 1919810;
	h ^= mix(v);
	h *= m;
	return mix(h);
}

__device__ uint64_t fasthash64_d(uint64_t v) {
	const uint64_t m = 0x880355f21e6d1965ULL;
	uint64_t h = 114514;
	h ^= mix(v);
	h *= m;
	return mix(h);
}
const int BUFSIZE = 1 << 20;
const int TABLESIZE = 100000007;
typedef std :: pair<uint64_t, data> pr;

__global__ void insert_kernel(uint64_t *hashtable, uint64_t *buf, int n){
  LIMITED_KERNEL_LOOP(i, n) {
    uint64_t key = buf[i];
    size_t slot = fasthash64_d(key) % TABLESIZE;
    while(1) {
      uint64_t prev = atomicCAS((unsigned long long int*)&hashtable[slot], 0ull, (unsigned long long int)key);
      if (prev == 0 || prev == key){
        buf[i] = slot;
        break;
      }
      slot = (slot + 1) % TABLESIZE;
    }
  }
}


uint64_t *a_d, *b_d;
uint64_t *a_buf_h, *b_buf_h, *a_buf_d, *b_buf_d;
bool *ab_buf;
data *table_h;
void init(){
  hipSetDevice(0);
  CHECK(hipMalloc(&a_d, sizeof(uint64_t) * TABLESIZE));
  hipSetDevice(1);
  CHECK(hipMalloc(&b_d, sizeof(uint64_t) * TABLESIZE));
  a_buf_h = (uint64_t*) malloc(sizeof(uint64_t) * BUFSIZE);
  b_buf_h = (uint64_t*) malloc(sizeof(uint64_t) * BUFSIZE);
  hipSetDevice(0);
  CHECK(hipMalloc(&a_buf_d, sizeof(uint64_t) * BUFSIZE));
  hipSetDevice(1);
  CHECK(hipMalloc(&b_buf_d, sizeof(uint64_t) * BUFSIZE));
  ab_buf = (bool*) malloc(sizeof(bool) * (BUFSIZE * 2));
  table_h = (data*) malloc(sizeof(data) * TABLESIZE * 2);
}

void insert(Loader<uint64_t> &keys, Loader<data> &vals){
  int cnt = 0, cnta = 0, cntb = 0;
  for(size_t i = 0; i < keys.count(); i++) {
    uint64_t v = keys.get();
    if (fasthash64(v) & 1) {
      a_buf_h[cnta++] = v;
      ab_buf[cnt++] = 0;
    } else {
      b_buf_h[cntb++] = v;
      ab_buf[cnt++] = 1;
    }
    if (cnta == BUFSIZE || cntb == BUFSIZE) {
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(uint64_t) * cnta, hipMemcpyHostToDevice));
      insert_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(uint64_t) * cntb, hipMemcpyHostToDevice));
      insert_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_h, a_buf_d, sizeof(uint64_t) * cnta, hipMemcpyDeviceToHost));
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_h, b_buf_d, sizeof(uint64_t) * cntb, hipMemcpyDeviceToHost));
      cnta = cntb = 0;
      for (int i = 0; i < cnt; i++) {
        if (ab_buf[i] == 0) {
          table_h[a_buf_h[cnta++]] = vals.get();
        } else {
          table_h[b_buf_h[cntb++] + TABLESIZE] = vals.get();
        }
      }
      cnt = cnta = cntb = 0;
    }
    
  }
  printf("qwq");
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(uint64_t) * cnta, hipMemcpyHostToDevice));
  insert_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(uint64_t) * cntb, hipMemcpyHostToDevice));
  insert_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_h, a_buf_d, sizeof(uint64_t) * cnta, hipMemcpyDeviceToHost));
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_h, b_buf_d, sizeof(uint64_t) * cnta, hipMemcpyDeviceToHost));
  cnta = cntb = 0;
  for (int i = 0; i < cnt; i++) {
    if (ab_buf[i] == 0) {
      table_h[a_buf_h[cnta++]] = vals.get();
    } else {
      table_h[b_buf_h[cntb++] + TABLESIZE] = vals.get();
    }
  }
  cnt = cnta = cntb = 0;
}



__global__ void find_kernel(const uint64_t *hashtable, uint64_t *buf, int n){
  LIMITED_KERNEL_LOOP(i, n) {
    uint64_t key = buf[i];
    size_t slot = fasthash64_d(key) % TABLESIZE;
    while(1) {
      if (hashtable[slot] == 0|| hashtable[slot] == key) {
        buf[i] = slot;
        break;
      }
      slot = (slot + 1) % TABLESIZE;
    }
  }
}
void find(Loader<uint64_t> &keys, Checker<data> &vals){
  int cnta = 0, cntb = 0, cnt = 0;
  for (size_t i = 0; i < keys.count(); i++) {
    uint64_t v = keys.get();
    if (fasthash64(v) & 1) {
      a_buf_h[cnta++] = v;
      ab_buf[cnt++] = 0;
    } else {
      b_buf_h[cntb++] = v;
      ab_buf[cnt++] = 1;
    }
    if (cnta == BUFSIZE || cntb == BUFSIZE ) {
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(uint64_t) * cnta, hipMemcpyHostToDevice));
      find_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(uint64_t) * cntb, hipMemcpyHostToDevice));
      find_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_h, a_buf_d, sizeof(uint64_t) * cnta, hipMemcpyDeviceToHost));
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_h, b_buf_d, sizeof(uint64_t) * cntb, hipMemcpyDeviceToHost));
      cnta = cntb = 0;
      for (int i = 0; i < cnt; i++) {
        if (ab_buf[i] == 0) {
          vals.put(table_h[a_buf_h[cnta++]]);
        }else{
          vals.put(table_h[b_buf_h[cntb++] + TABLESIZE]);
        }
      }
      cnta = cntb = cnt = 0;
    }
  }
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(uint64_t) * cnta, hipMemcpyHostToDevice));
  find_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(uint64_t) * cntb, hipMemcpyHostToDevice));
  find_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_h, a_buf_d, sizeof(uint64_t) * cnta, hipMemcpyDeviceToHost));
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_h, b_buf_d, sizeof(uint64_t) * cntb, hipMemcpyDeviceToHost));
  cnta = cntb = 0;
  for (int i = 0; i < cnt; i++) {
    if (ab_buf[i] == 0) {
      vals.put(table_h[a_buf_h[cnta++]]);
    }else{
      vals.put(table_h[b_buf_h[cntb++] + TABLESIZE]);
    }
  }
}


void clear(){
  free(a_buf_h);
  free(b_buf_h);
  free(ab_buf);
  free(table_h);
  CHECK(hipFree(a_d));
  CHECK(hipFree(b_d));
  CHECK(hipFree(a_buf_d));
  CHECK(hipFree(b_buf_d));
}