#include "hip/hip_runtime.h"
#include "dinner123.h"
#include "hash_table.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <map>
size_t size(){
  return 0;
}
size_t capacity(){
  return 0;
}
const int MultiProcessorCount = 80;
#define mix(h) ({					\
			(h) ^= (h) >> 23;		\
			(h) *= 0x2127599bf4325c37ULL;	\
			(h) ^= (h) >> 47; })


uint64_t fasthash64(uint64_t v) {
	const uint64_t m = 0x880355f21e6d1965ULL;
	uint64_t h = 1919810;
	h ^= mix(v);
	h *= m;
	return mix(h);
}

__device__ uint64_t fasthash64_d(uint64_t v) {
	const uint64_t m = 0x880355f21e6d1965ULL;
	uint64_t h = 114514;
	h ^= mix(v);
	h *= m;
	return mix(h);
}
const int BUFSIZE = 1 << 14;
const int TABLESIZE = 100000007;
typedef std :: pair<uint64_t, data> pr;

__global__ void insert_kernel(pr *hashtable, const pr *buf, int n){
  LIMITED_KERNEL_LOOP(i, n) {
    uint64_t key = buf[i].first;
    data val = buf[i].second;
    size_t slot = fasthash64_d(key) % TABLESIZE;
    while(1) {
      uint64_t prev = atomicCAS((unsigned long long int*)&hashtable[slot].first, 0ull, (unsigned long long int)key);
      if (prev == 0 || prev == key) {
        hashtable[slot].second = val;
        break;
      }
      slot = (slot + 1) % TABLESIZE;
    }
  }
}


pr *a_d, *b_d;
pr *a_buf_h, *b_buf_h, *a_buf_d, *b_buf_d;
bool *ab_buf;

void init(){
  hipSetDevice(0);
  CHECK(hipMalloc(&a_d, sizeof(pr) * TABLESIZE));
  hipSetDevice(1);
  CHECK(hipMalloc(&b_d, sizeof(pr) * TABLESIZE));
  a_buf_h = (pr*) malloc(sizeof(pr) * BUFSIZE);
  b_buf_h = (pr*) malloc(sizeof(pr) * BUFSIZE);
  hipSetDevice(0);
  CHECK(hipMalloc(&a_buf_d, sizeof(pr) * BUFSIZE));
  hipSetDevice(1);
  CHECK(hipMalloc(&b_buf_d, sizeof(pr) * BUFSIZE));
  ab_buf = (bool*) malloc(sizeof(bool) * (BUFSIZE * 2));
}

void insert(Loader<uint64_t> &keys, Loader<data> &vals){
  
  int cnta = 0, cntb = 0;
  for(size_t i = 0; i < keys.count(); i++) {
    uint64_t v = keys.get();
    if (fasthash64(v) & 1) {
      a_buf_h[cnta++] = std :: make_pair(v, vals.get());
    } else {
      b_buf_h[cntb++] = std :: make_pair(v, vals.get());
    }
    if (cnta == BUFSIZE) {
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(pr) * cnta, hipMemcpyHostToDevice));
      insert_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
      cnta = 0;
    }
    if (cntb == BUFSIZE) {
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(pr) * cntb, hipMemcpyHostToDevice));
      insert_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
      cntb = 0;
    }
  }
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(pr) * cnta, hipMemcpyHostToDevice));
  insert_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
  
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(pr) * cntb, hipMemcpyHostToDevice));
  insert_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
  
}



__global__ void find_kernel(const pr *hashtable, pr *buf, int n){
  LIMITED_KERNEL_LOOP(i, n) {
    uint64_t key = buf[i].first;
    size_t slot = fasthash64_d(key) % TABLESIZE;
    while(1) {
      if (hashtable[slot].first == 0|| hashtable[slot].first == key) {
        buf[i].second = hashtable[slot].second;
        break;
      }
      slot = (slot + 1) % TABLESIZE;
    }
  }
}
void find(Loader<uint64_t> &keys, Checker<data> &vals){
  int cnta = 0, cntb = 0, cnt = 0;
  for (size_t i = 0; i < keys.count(); i++) {
    uint64_t v = keys.get();
    if (fasthash64(v) & 1) {
      a_buf_h[cnta++].first = v;
      ab_buf[cnt++] = 0;
    } else {
      b_buf_h[cntb++].first = v;
      ab_buf[cnt++] = 1;
    }
    if (cnta == BUFSIZE || cntb == BUFSIZE ) {
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(pr) * cnta, hipMemcpyHostToDevice));
      find_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(pr) * cntb, hipMemcpyHostToDevice));
      find_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
      hipSetDevice(0);
      CHECK(hipMemcpy(a_buf_h, a_buf_d, sizeof(pr) * cnta, hipMemcpyDeviceToHost));
      hipSetDevice(1);
      CHECK(hipMemcpy(b_buf_h, b_buf_d, sizeof(pr) * cntb, hipMemcpyDeviceToHost));
      cnta = cntb = 0;
      for (int i = 0; i < cnt; i++) {
        if (ab_buf[i] == 0) {
          vals.put(a_buf_h[cnta++].second);
        }else{
          vals.put(b_buf_h[cntb++].second);
        }
      }
      cnta = cntb = cnt = 0;
    }
  }
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_d, a_buf_h, sizeof(pr) * cnta, hipMemcpyHostToDevice));
  find_kernel<<<MultiProcessorCount, 1024>>>(a_d, a_buf_d, cnta);
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_d, b_buf_h, sizeof(pr) * cntb, hipMemcpyHostToDevice));
  find_kernel<<<MultiProcessorCount, 1024>>>(b_d, b_buf_d, cntb);
  hipSetDevice(0);
  CHECK(hipMemcpy(a_buf_h, a_buf_d, sizeof(pr) * cnta, hipMemcpyDeviceToHost));
  hipSetDevice(1);
  CHECK(hipMemcpy(b_buf_h, b_buf_d, sizeof(pr) * cntb, hipMemcpyDeviceToHost));
  cnta = cntb = 0;
  for (int i = 0; i < cnt; i++) {
    if (ab_buf[i] == 0) {
      vals.put(a_buf_h[cnta++].second);
    }else{
      vals.put(b_buf_h[cntb++].second);
    }
  }
}


void clear(){
  free(a_buf_h);
  free(b_buf_h);
  free(ab_buf);
  CHECK(hipFree(a_d));
  CHECK(hipFree(b_d));
  CHECK(hipFree(a_buf_d));
  CHECK(hipFree(b_buf_d));
}