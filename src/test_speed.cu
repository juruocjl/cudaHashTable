#include "hip/hip_runtime.h"
#include "reader.h"
#include "v5.cuh"
#include <bits/stdc++.h>
#include "dinner123.h"
#include <hip/hip_runtime.h>
using namespace std;
HashTable<uint64_t, float, 64> gpuhstb;

template <typename T>
double GPU_PERF(T func) {
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	func();
	hipEventRecord(stop);
	CHECK(hipStreamSynchronize(0));
	float duration = 0;
	hipEventElapsedTime(&duration, start, stop);
	return duration / 1000;
}
int main() {
	file_loader<uint64_t> insertion("/root/cudaHashTable/data/akdream1/A/part0.keys");
	file_loader<uint64_t> finding("/root/cudaHashTable/data/akdream1/A/part0.keys");
	//data_loader insertion("/root/cudaHashTable/data/sample/A/part_0");
	//file_loader<uint64_t> finding("/root/cudaHashTable/data/sample/A/part_0.keys");
	cerr << "load use " << 1. * clock() / CLOCKS_PER_SEC << endl;
	int n = insertion.count();
	cerr << "n = " << n << endl;
	uint64_t *answer_gpu;// = (uint64_t *)malloc(finding.count() * sizeof(uint64_t));
	CHECK(hipHostAlloc(&answer_gpu, finding.count() * sizeof(uint64_t), hipHostMallocMapped));
	bool  *exist_gpu;// = (bool *)malloc(finding.count() * sizeof(uint64_t));
	CHECK(hipHostAlloc(&exist_gpu, finding.count() * sizeof(bool), hipHostMallocMapped));

	double gpu_insert_time = GPU_PERF([&] { gpuhstb.insert(n, insertion.data(), 114514); });
	double gpu_insert_qps = n / gpu_insert_time;
	cerr << "insert_time=" << gpu_insert_time << endl;
	cerr << "insert_qps=" << gpu_insert_qps << endl;
	double gpu_find_time = GPU_PERF([&] { gpuhstb.find(n, finding.data(), answer_gpu, exist_gpu); });
	double gpu_find_qps = n / gpu_find_time;
	cerr << "find_time=" << gpu_find_time << endl;
	cerr << "find_qps=" << gpu_find_qps << endl;
	return 0;
}